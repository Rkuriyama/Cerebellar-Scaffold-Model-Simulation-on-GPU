#include "hip/hip_runtime.h"
#include "UserInputFunctions.h"

__device__ char PoissonProcess (const CTYPE r, const CTYPE time, const CTYPE freq, const CTYPE start){
	return ( r < freq/sec*DT );
}

__device__ char PeriodicFiring (const float r, const CTYPE time, const CTYPE freq, const CTYPE start){
	return (time == start) || ( ((int)((time - start - DT)*freq/sec)) != ((int)((time - start)*freq/sec)) );
}

__device__ CTYPE SinusoidalOscillation (const float max, const float mean, const float osci, const float shift, const CTYPE time){
    return max/2 * (1 - cosf( 2*M_PI*(time/sec)/osci + shift ));
}


__device__ char sustained_input(const float r, const CTYPE time){
    char flag = 0;
    flag = ( ( 200.0 <= time && time < 1200.0 ) && PoissonProcess(r, time, 30.0, 200.0) ) || ( ( ( 0.0 <= time && time < 200.0) || ( 1200 <= time ) ) && PoissonProcess(r, time, 5.0, 0.0) );
    //flag = PoissonProcess(r, time, 5.0, 0.0);
    return (flag)?1:0;
}

__device__ char transient_input(const float r, const CTYPE time){
    char flag = 0;
    flag = ( ( 200.0 <= time && time < 205.0 ) && PoissonProcess(r, time, 200.0, 200.0) ) || ( ( ( 0.0 <= time && time < 200.0) || ( 205.0 <= time ) ) && PoissonProcess(r, time, 5.0, 0.0) );
    //flag = ( ( 200.0 <= time && time < 205.0 ) && PeriodicFiring(r, time, 200.0, 200.0) ) || ( ( ( 0.0 <= time && time < 200.0) || ( 205.0 <= time ) ) && PoissonProcess(r, time, 5.0, 0.0) );
    //flag = ( (   0.0 <= time ) && PoissonProcess(r, time, 50.0, 0.0) );

    //flag = PoissonProcess(r, time, 5.0, 0.0);
    return (flag)?1:0;
}


__host__ void InitInputStimulation( InputFunctionsStruct *List, Neuron *host_neurons, int *NeuronTypeID){
	FILE *fp;
	char str[256] = {'\0'};
	unsigned int *host_Id_list;
	int i;


	if ( (fp = fopen( "transient_input.dat" , "r")) == NULL){
		fprintf(stderr, "cannot open file: transient_input.dat\n");
		exit(1);
	}

	List[0].type = NeuronTypeID[glomerulus];
	List[0].base_id = host_neurons[NeuronTypeID[glomerulus]].base_id;
	List[0].num = 204800;
	List[0].func_id = 0;
	i=0;
	List[0].IdList = (unsigned int*)malloc(sizeof(unsigned int)*List[0].num);
	while( fgets(str, 256, fp) != NULL ){
		sscanf(str, "%u", &List[0].IdList[i]);
		i++;
	}
	fclose(fp);

	if ( (fp = fopen( "sustained_input.dat" , "r")) == NULL){
		fprintf(stderr, "cannot open file: sustained_input.dat\n");
		exit(1);
	}

	List[1].type = NeuronTypeID[glomerulus];
	List[1].base_id = host_neurons[NeuronTypeID[glomerulus]].base_id;
	List[1].num = 204800;
	List[1].func_id = 1;
	i=0;
	List[1].IdList = (unsigned int*)malloc(sizeof(unsigned int)*List[1].num);
	while( fgets(str, 256, fp) != NULL ){
		sscanf(str, "%u", &List[1].IdList[i]);
		i++;
	}
	fclose(fp);

    return;
}
//__device__ pointFunction_t d_pInputFunctions[] = {background_noise, tone_stim, puff_stim};
__device__ pointFunction_t d_pInputFunctions[] = {transient_input, sustained_input};

__global__ void InputStimulation( const int n, char *spike,
                                  hiprandStatePhilox4_32_10_t *state,
                                  const int num, const int base_id,
                                  const unsigned int *IdList,
                                  const int target_row, const int total_nn,
	       			  int func_id){
	// cindicesはここでは使わない pallot neuron
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	if( i < num ){
		//Glomのnoise発火
		int global_id = base_id + IdList[i];
		float r[STEP_MAX];
		float4 tmp_r;

		for(int step = 0; step < STEP_MAX; step++){
			switch(step%4){
				case 0:
					tmp_r = hiprand_uniform4(&state[i]);
					r[step] = tmp_r.x;
					break;
				case 1: r[step] = tmp_r.y; break;
				case 2: r[step] = tmp_r.z; break;
				case 3: r[step] = tmp_r.w; break;
			}
		}

		char spike_ = 0;
		for(int step = 0; step < STEP_MAX;step++){
			spike_ += d_pInputFunctions[func_id](r[step], (CTYPE)n +((CTYPE)step)*DT);
		}
		spike[target_row + global_id] = (spike_)? 1 : 0;
	}
	return;
}
