#include "hip/hip_runtime.h"
#include "init_params.h"
#include "option.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

FILE *wout = fopen("weight.dat","w");

int LoadConnectivityFile(const char *file_name,unsigned int **host_rptr, unsigned int **d_rptr, unsigned int **d_cindices, CTYPE **d_val, CTYPE weight ,int PreSN_num,int PostSN_num){
	PreSN_num = (PreSN_num < 1)? 1: PreSN_num;
	PostSN_num = (PostSN_num < 1)? 1000: PostSN_num;


	FILE *fp;
	if((fp = fopen( file_name ,"r")) == NULL ){
		fprintf(stderr, "can't open file :  %s\n",file_name);
		exit(1);
	}

	weight = (weight < 0)?(-1)*weight: weight;
	int num_of_data = PostSN_num*10;
	unsigned int *rptr = NULL;
	unsigned int *cindices = NULL;
	CTYPE *val =NULL;

	int max_conv = 0;


	rptr = (unsigned int *)malloc( (PostSN_num+1)*sizeof(unsigned int) );
	cindices = (unsigned int *)malloc( num_of_data*sizeof(unsigned int) );
	val = (CTYPE *)malloc( num_of_data*sizeof(CTYPE) );

	if(rptr == NULL || cindices == NULL || val == NULL){
		fprintf(stderr,"malloc error\n");
		exit(1);
	}

	char str[256] = {'\0'};
	int i = 0;
	int prev_post_id = 0;
	int post_id;
	rptr[0] = 0;
	while( fgets(str, 256, fp) != NULL ){

		//sscanf(str, "%d %d %f", &cindices[i], &post_id, &val[i] );
		sscanf(str, "%d %d", &cindices[i], &post_id );
		
		val[i] = 1;

		// 本来はpost_id > prev_post_id (ソート済み前提)
		if(post_id != prev_post_id) {
			for(int j=prev_post_id+1;j<post_id+1;j++) rptr[j] = i;
			prev_post_id = post_id;
		}
		i++;

		// 拡張
		if(i > num_of_data-1){
			float avg = (post_id != 0)?(float)i/(float)(post_id):i;
			num_of_data = (int)(avg*PostSN_num);

			//fprintf(stderr, "realloc phase %d to %d\n", i, num_of_data);

			unsigned int *i_tmp=NULL;
			CTYPE *c_tmp=NULL;
			if(( i_tmp = (unsigned int *)realloc(cindices, num_of_data*sizeof(unsigned int))) == NULL){
				free(cindices);
				exit(1);
			}else{
				if(cindices != i_tmp){
					cindices = i_tmp;
				}
			}

			if(( c_tmp = (CTYPE *)realloc(val, num_of_data*sizeof(CTYPE) )) == NULL){
				free(val);
				exit(1);
			}else{
				if(val != c_tmp){
					val = c_tmp;
				}
			}

		}
	}

	if(num_of_data != i){
		num_of_data = i;
		for(int j = post_id+1; j < PostSN_num+1;j++){
			rptr[j] = num_of_data;
		}

		// 縮小
		//fprintf(stderr, "realloc phase :to %d\n", num_of_data);
		unsigned int *i_tmp = NULL;
		CTYPE *c_tmp = NULL;
		if(( i_tmp = (unsigned int *)realloc(cindices, num_of_data*sizeof(unsigned int))) == NULL){
			fprintf(stderr, "can't realloc memory in roading phase: %s\n", file_name);
			free(cindices);
			exit(1);
		}else{
			if(cindices != i_tmp)cindices = i_tmp;
		}

		if(( c_tmp = (CTYPE *)realloc(val, num_of_data*sizeof(CTYPE))) == NULL){
			fprintf(stderr, "can't realloc memory in roading phase: %s\n", file_name);
			free(val);
			exit(1);
		}else{
			if(val != c_tmp) val = c_tmp;
		}
	}

	for(int i = 0; i < PostSN_num; i++) max_conv = (max_conv < rptr[i+1]-rptr[i])?rptr[i+1]-rptr[i]:max_conv;

	CUDA_SAFE_CALL( hipMalloc( d_rptr, sizeof(unsigned int)*(PostSN_num+1)) );
	CUDA_SAFE_CALL( hipMalloc( d_cindices, sizeof(unsigned int)*num_of_data) );
	CUDA_SAFE_CALL( hipMalloc( d_val, sizeof(CTYPE)*num_of_data));

	CUDA_SAFE_CALL( hipMemcpy( *d_rptr, rptr, sizeof(unsigned int)*(PostSN_num+1), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( *d_cindices, cindices, sizeof(unsigned int)*num_of_data, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( *d_val, val, sizeof(CTYPE)*num_of_data, hipMemcpyHostToDevice));

	*host_rptr = rptr;

	fclose(fp);
	//free(rptr);
	free(cindices);
	free(val);

	return max_conv;
}

int set_neuron_params(Neuron *n,enum NeuronType type,const char* filename, char duplicate, int num, int base_id, CTYPE Cm, CTYPE tau_m, CTYPE El, CTYPE dt_ref, CTYPE Ie, CTYPE Vr, CTYPE Vth, CTYPE tau_exc, CTYPE tau_inh, CTYPE gL, enum DeviceType device_type  ){
	static int CPU = 0;
	static int GPU = 0;
	int target = 0;
	if( device_type == NORMAL ){
		target = GPU;
		GPU++;
	} else {
	       	target = (int)TotalNumOfCellTypes - 1 - CPU;
		CPU++;
	}
	n[target].type = target;
	strcpy(n[target].filename, filename);
	n[target].num = num;
	n[target].base_id = base_id;
	n[target].Cm = Cm;
	n[target].tau_m = tau_m;
	n[target].El = El;
	n[target].dt_ref = dt_ref;
	n[target].Ie = Ie;
	n[target].Vr = Vr;
	n[target].Vth = Vth;
	n[target].tau_exc = tau_exc;
	n[target].tau_inh = tau_inh;
	n[target].gL = gL;
	n[target].duplicate = duplicate;
	n[target].dev_type = device_type;
	return target;
}
int set_connectivity_params(Connectivity *c, Neuron *neurons, enum ConnectionType type,const char*filename, int preNum, int postNum, int preType, int postType, CTYPE initial_weight, CTYPE delay, int UseParallelReduction  ){
	static int GPU = 0;
	static int CPU = 0;
	int target = 0;

	if ( neurons[postType].dev_type == OUTPUT ){
		target = (int)TotalNumOfConnectivityTypes - 1 - CPU;
		CPU++;
	}else{
		target = GPU;
		GPU++;
	}

	c[target].type = type;
	c[target].preNum = preNum;
	c[target].postNum = postNum;
	c[target].preType = preType;
	c[target].postType = postType;
	c[target].initial_weight = initial_weight;
	c[target].delay = delay;
	c[target].max_conv = LoadConnectivityFile(filename,&c[target].host_rptr, &c[target].rptr, &c[target].cindices, &c[target].val,initial_weight, preNum, postNum );
	c[target].pr = (UseParallelReduction);


	return target;
}

int set_base_id(Neuron *Neurons){
	int base = 0;
	for(int i = 0;i < TotalNumOfCellTypes;i++){
		Neurons[i].base_id = base;
		base += Neurons[i].num;
        fprintf(stderr, "%s: %d\n", Neurons[i].filename, base);
	}
	return base;
}

__global__ void InitParams( CTYPE *u, CTYPE *g_exc, CTYPE *dg_exc, CTYPE *g_inh, CTYPE *dg_inh, int *refractory_time_left, char *spike , Neuron *Neurons ,char *type, hiprandStatePhilox4_32_10_t *state, const int total_nn){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if( i < total_nn){
		u[i] = Neurons[type[i]].Vr + (Neurons[type[i]].Vth - Neurons[type[i]].Vr)*hiprand_uniform(&state[i]);
		g_exc[i] = 0.f;
		dg_exc[i] = 0.f;
		g_inh[i] = 0.f;
		dg_inh[i] = 0.f;
		refractory_time_left[i] = 0;
		spike[i] = 0;
	}
};

__host__ void Host_InitParams( CTYPE *u, CTYPE *g_exc, CTYPE *dg_exc, CTYPE *g_inh, CTYPE *dg_inh, int *refractory_time_left, char *spike , Neuron *Neurons ,char *type, const int total_nn){
	srand( (unsigned)time(NULL) );
	for(int i = 0 ; i < total_nn; i++){
		u[i] = Neurons[type[i]].Vr + (Neurons[type[i]].Vth - Neurons[type[i]].Vr)*(  (rand() + 1.0)/(2.0 + RAND_MAX)  );
		g_exc[i] = 0.f;
		dg_exc[i] = 0.f;
		g_inh[i] = 0.f;
		dg_inh[i] = 0.f;
		refractory_time_left[i] = 0;
		spike[i] = 0;
	}
};


void init_neurons_params( Neuron *Neurons, int *NeuronTypeID){

	NeuronTypeID[granule_cell] = set_neuron_params(
		Neurons,
		granule_cell,
		"granule_cell.dat",
		0,
		367626,
		36323,
		3.0,
		2.,
		-74.0,
		1.5,
		0.0,
		-84.0,
		-42.0,
		0.5,
		10.,
		0.29,
		NORMAL
	);

	NeuronTypeID[glomerulus] = set_neuron_params(
		Neurons,
		glomerulus,
		"glomerulus.dat",
		0,
		28798,
		7525,
		0,
		0,
		0,
		0,
		0,
		0,
		0,
		0,
		0,
		0,
		INPUT
	);

	NeuronTypeID[purkinje_cell] = set_neuron_params(
		Neurons,
		purkinje_cell,
		"purkinje_cell.dat",
		0,
		289,
		161,
		620.0,
		88.,
		-62.0,
		0.8,
		600.0,
		-72.0,
		-47.0,
		0.5,
		1.6,
		7.1,
		NORMAL
	);

	NeuronTypeID[golgi_cell] = set_neuron_params(
		Neurons,
		golgi_cell,
		"golgi_cell.dat",
		0,
		864,
		450,
		76.0,
		21.,
		-65.0,
		2.0,
		36.8,
		-75.0,
		-55.0,
		0.5,
		10.0,
		3.3,
		NORMAL
	);

	NeuronTypeID[stellate_cell] = set_neuron_params(
		Neurons,
		stellate_cell,
		"stellate_cell.dat",
		0,
		2396,
		2753,
		14.6,
		14.6,
		-68.0,
		1.6,
		15.6,
		-78.0,
		-53.0,
		0.64,
		2.0,
		1.6,
		NORMAL
	);

	NeuronTypeID[basket_cell] = set_neuron_params(
		Neurons,
		basket_cell,
		"basket_cell.dat",
		0,
		2376,
		5149,
		14.6,
		14.6,
		-68.0,
		1.6,
		15.6,
		-78.0,
		-53.0,
		0.64,
		2.0,
		1.6,
		NORMAL
	);

	NeuronTypeID[dcn_cell] = set_neuron_params(
		Neurons,
		dcn_cell,
		"dcn_cell.dat",
		0,
		57,
		47,
		89.0,
		57.0,
		-59.0,
		3.7,
		55.8,
		-69.0,
		-48.0,
		7.1,
		13.6,
		4.3,
		OUTPUT
	);


	NeuronTypeID[dcn_interneuron] = set_neuron_params(
		Neurons,
		dcn_interneuron,
		"dcn_interneuron.dat",
		0,
		57,
		104,
		56.0,
		56.0,
		-40.0,
		0.8,
		7.0,
		-55.0,
		-39.0,
		3.64,
		1.14,
		1.0,
		OUTPUT
	);

	NeuronTypeID[io_cell] = set_neuron_params(
		Neurons,
		io_cell,
		"io_cell.dat",
		1,
		47,
		0,
		189.0,
		11.0011641443539,
		-45.0,
		1.0,
		0.0,
		-45.0,
		-35.0,
		1.0,
		60.0,
		17.18,
		INPUT
	);

}

void init_connectivity_params(Connectivity *connectivities, Neuron *neurons, int *NeuronTypeID, int *ConnectivityTypeID){

	ConnectivityTypeID[parallel_fiber_to_purkinje] = set_connectivity_params(
		connectivities,
		neurons,
		parallel_fiber_to_purkinje,
		"parallel_fiber_to_purkinje.dat",
		367626,
		289,
		NeuronTypeID[granule_cell],
		NeuronTypeID[purkinje_cell],
		0.02,//0.007,
		5.0,
		1
	);

	ConnectivityTypeID[parallel_fiber_to_basket] = set_connectivity_params(
		connectivities,
		neurons,
		parallel_fiber_to_basket,
		"parallel_fiber_to_basket.dat",
		367626,
		2376,
		NeuronTypeID[granule_cell],
		NeuronTypeID[basket_cell],
		0.2,
		5.0,
		0
	);

	ConnectivityTypeID[parallel_fiber_to_stellate] = set_connectivity_params(
		connectivities,
		neurons,
		parallel_fiber_to_stellate,
		"parallel_fiber_to_stellate.dat",
		367626,
		2396,
		NeuronTypeID[granule_cell],
		NeuronTypeID[stellate_cell],
		0.2,
		5.0,
		0
	);

	ConnectivityTypeID[parallel_fiber_to_golgi] = set_connectivity_params(
		connectivities,
		neurons,
		parallel_fiber_to_golgi,
		"parallel_fiber_to_golgi.dat",
		367626,
		864,
		NeuronTypeID[granule_cell],
		NeuronTypeID[golgi_cell],
		0.4,
		5.0,
		0
	);

	ConnectivityTypeID[glomerulus_to_dcn] = set_connectivity_params(
		connectivities,
		neurons,
		glomerulus_to_dcn,
		"glomerulus_to_dcn.dat",
		28798,
		57,
		NeuronTypeID[glomerulus],
		NeuronTypeID[dcn_cell],
		0.006,
		3.0,
		0
	);

	ConnectivityTypeID[ascending_axon_to_golgi] = set_connectivity_params(
		connectivities,
		neurons,
		ascending_axon_to_golgi,
		"ascending_axon_to_golgi.dat",
		367626,
		864,
		NeuronTypeID[granule_cell],
		NeuronTypeID[golgi_cell],
		20.,
		5.0,
		0
	);

	ConnectivityTypeID[ascending_axon_to_purkinje] = set_connectivity_params(
		connectivities,
		neurons,
		ascending_axon_to_purkinje,
		"ascending_axon_to_purkinje.dat",
		367626,
		289,
		NeuronTypeID[granule_cell],
		NeuronTypeID[purkinje_cell],
		75.0,
		2.0,
		0
	);

	ConnectivityTypeID[glomerulus_to_golgi] = set_connectivity_params(
		connectivities,
		neurons,
		glomerulus_to_golgi,
		"glomerulus_to_golgi.dat",
		28798,
		864,
		NeuronTypeID[glomerulus],
		NeuronTypeID[golgi_cell],
		2.0,
		4.0,
		0
	);

	ConnectivityTypeID[golgi_to_golgi] = set_connectivity_params(
		connectivities,
		neurons,
		golgi_to_golgi,
		"golgi_to_golgi.dat",
		864,
		864,
		NeuronTypeID[golgi_cell],
		NeuronTypeID[golgi_cell],
		-8.0,
		1.0,
		0
	);

	ConnectivityTypeID[io_to_dcn] = set_connectivity_params(
		connectivities,
		neurons,
		io_to_dcn,
		"io_to_dcn.dat",
		47,
		57,
		NeuronTypeID[io_cell],
		NeuronTypeID[dcn_cell],
		0.1,
		4.0,
		0
	);

	ConnectivityTypeID[io_to_dcn_interneuron] = set_connectivity_params(
		connectivities,
		neurons,
		io_to_dcn_interneuron,
		"io_to_dcn_interneuron.dat",
		47,
		57,
		NeuronTypeID[io_cell],
		NeuronTypeID[dcn_interneuron],
		0.2,
		5.0,
		0
	);

	ConnectivityTypeID[purkinje_to_dcn] = set_connectivity_params(
		connectivities,
		neurons,
		purkinje_to_dcn,
		"purkinje_to_dcn.dat",
		289,
		57,
		NeuronTypeID[purkinje_cell],
		NeuronTypeID[dcn_cell],
		-0.0075,
		4.0,
		0
	);

	ConnectivityTypeID[basket_to_purkinje] = set_connectivity_params(
		connectivities,
		neurons,
		basket_to_purkinje,
		"basket_to_purkinje.dat",
		2376,
		289,
		NeuronTypeID[basket_cell],
		NeuronTypeID[purkinje_cell],
		-9.0,
		4.0,
		0
	);

	ConnectivityTypeID[stellate_to_purkinje] = set_connectivity_params(
		connectivities,
		neurons,
		stellate_to_purkinje,
		"stellate_to_purkinje.dat",
		2396,
		289,
		NeuronTypeID[stellate_cell],
		NeuronTypeID[purkinje_cell],
		-8.5,
		5.0,
		0
	);

	ConnectivityTypeID[stellate_to_stellate] = set_connectivity_params(
		connectivities,
		neurons,
		stellate_to_stellate,
		"stellate_to_stellate.dat",
		2396,
		2396,
		NeuronTypeID[stellate_cell],
		NeuronTypeID[stellate_cell],
		-2.0,
		1.0,
		0
	);

	ConnectivityTypeID[basket_to_basket] = set_connectivity_params(
		connectivities,
		neurons,
		basket_to_basket,
		"basket_to_basket.dat",
		2376,
		2376,
		NeuronTypeID[basket_cell],
		NeuronTypeID[basket_cell],
		-2.5,
		1.0,
		0
	);

	ConnectivityTypeID[glomerulus_to_granule] = set_connectivity_params(
		connectivities,
		neurons,
		glomerulus_to_granule,
		"glomerulus_to_granule.dat",
		28798,
		367626,
		NeuronTypeID[glomerulus],
		NeuronTypeID[granule_cell],
		9.0,
		4.0,
		0
	);

	ConnectivityTypeID[golgi_to_granule] = set_connectivity_params(
		connectivities,
		neurons,
		golgi_to_granule,
		"golgi_to_granule.dat",
		864,
		367626,
		NeuronTypeID[golgi_cell],
		NeuronTypeID[granule_cell],
		-5.0,
		2.0,
		0
	);

	ConnectivityTypeID[io_to_basket] = set_connectivity_params(
		connectivities,
		neurons,
		io_to_basket,
		"io_to_basket.dat",
		47,
		2376,
		NeuronTypeID[io_cell],
		NeuronTypeID[basket_cell],
		1.0,
		70.0,
		0
	);

	ConnectivityTypeID[io_to_stellate] = set_connectivity_params(
		connectivities,
		neurons,
		io_to_stellate,
		"io_to_stellate.dat",
		47,
		2396,
		NeuronTypeID[io_cell],
		NeuronTypeID[stellate_cell],
		1.0,
		70.0,
		0
	);

	ConnectivityTypeID[io_to_purkinje] = set_connectivity_params(
		connectivities,
		neurons,
		io_to_purkinje,
		"io_to_purkinje.dat",
		47,
		289,
		NeuronTypeID[io_cell],
		NeuronTypeID[purkinje_cell],
		350.0,
		4.0,
		0
	);
}

